#include "hip/hip_runtime.h"
// File: src/ct_projector_3d.cu
//
// 2025-04-30  – changes:
//   • all kernels now expect **M_inv** (row-major, 9 floats)
//   • apply_affine_inverse_3d no longer inverts – it just multiplies by M_inv
//   • midpoint-to-voxel index uses IEEE-754 round-to-nearest-even
//   • host wrappers renamed arguments accordingly (but keep same function names
//     so the Python extension API remains unchanged)

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <cmath>
#include <cstdio>

#define THREADS 1024

// -----------------------------------------------
// HELPER: apply_affine_inverse_3d
//   (x,y,z) → (i,j,k)   using pre-computed M_inv
// -----------------------------------------------
__device__ __forceinline__
void apply_affine_inverse_3d(
    float x, float y, float z,
    const float* M_inv,   // [9] = a11 … a33  (row-major)
    const float* b,       // [3]
    float& i_out, float& j_out, float& k_out)
{
    float x_shift = x - b[0];
    float y_shift = y - b[1];
    float z_shift = z - b[2];

    // layout: [0 1 2 ; 3 4 5 ; 6 7 8]
    i_out = M_inv[0]*x_shift + M_inv[1]*y_shift + M_inv[2]*z_shift;
    j_out = M_inv[3]*x_shift + M_inv[4]*y_shift + M_inv[5]*z_shift;
    k_out = M_inv[6]*x_shift + M_inv[7]*y_shift + M_inv[8]*z_shift;
}

// (the three kernels are unchanged except that the argument name is M_inv
//  and the rounding now uses __float2int_rn.)

// ……………………… compute_intersections_3d_kernel ………………………
__global__ void compute_intersections_3d_kernel(
    int n_x, int n_y, int n_z,
    const float* src_xyz, const float* dst_xyz, int n_ray,
    const float* M_inv, const float* b,
    float* t_out)
{
    int ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (ray_id >= n_ray) return;

    float sx = src_xyz[3*ray_id+0], sy = src_xyz[3*ray_id+1], sz = src_xyz[3*ray_id+2];
    float dx = dst_xyz[3*ray_id+0], dy = dst_xyz[3*ray_id+1], dz = dst_xyz[3*ray_id+2];

    float si, sj, sk, di, dj, dk;
    apply_affine_inverse_3d(sx,sy,sz, M_inv,b, si,sj,sk);
    apply_affine_inverse_3d(dx,dy,dz, M_inv,b, di,dj,dk);

    float dirx = di-si, diry = dj-sj, dirz = dk-sk;
    int n_int = (n_x+1)+(n_y+1)+(n_z+1);
    float* t_vals = &t_out[ray_id*n_int];

    int cnt = 0;
    // x-planes
    for (int i=0;i<=n_x;++i){
        float plane = float(i)-0.5f;
        if (fabsf(dirx)<1e-12f) t_vals[cnt++] = INFINITY;
        else{
            float tt = (plane-si)/dirx;
            t_vals[cnt++] = (tt<0.f||tt>1.f)?INFINITY:tt;
        }
    }
    // y-planes
    for (int j=0;j<=n_y;++j){
        float plane = float(j)-0.5f;
        if (fabsf(diry)<1e-12f) t_vals[cnt++] = INFINITY;
        else{
            float tt = (plane-sj)/diry;
            t_vals[cnt++] = (tt<0.f||tt>1.f)?INFINITY:tt;
        }
    }
    // z-planes
    for (int k=0;k<=n_z;++k){
        float plane = float(k)-0.5f;
        if (fabsf(dirz)<1e-12f) t_vals[cnt++] = INFINITY;
        else{
            float tt = (plane-sk)/dirz;
            t_vals[cnt++] = (tt<0.f||tt>1.f)?INFINITY:tt;
        }
    }
    // bubble sort (unchanged)
    for (int i=0;i<n_int-1;++i)
        for (int j=0;j<n_int-1-i;++j)
            if (t_vals[j]>t_vals[j+1]){
                float tmp=t_vals[j]; t_vals[j]=t_vals[j+1]; t_vals[j+1]=tmp;
            }
}

// ……………………… forward_project_3d_kernel ………………………
__global__ void forward_project_3d_kernel(
    const float* volume,int batch,int n_x,int n_y,int n_z,
    const float* t_sorted,int n_int,
    const float* src_xyz,const float* dst_xyz,int n_ray,
    const float* M_inv,const float* b,
    float* out)
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int total = batch*n_ray;
    if (gid>=total) return;

    int b_idx = gid / n_ray;
    int r_idx = gid % n_ray;

    float sx = src_xyz[3*r_idx+0], sy = src_xyz[3*r_idx+1], sz = src_xyz[3*r_idx+2];
    float dx = dst_xyz[3*r_idx+0], dy = dst_xyz[3*r_idx+1], dz = dst_xyz[3*r_idx+2];
    float vx=dx-sx, vy=dy-sy, vz=dz-sz;

    const float* t_vals = &t_sorted[r_idx*n_int];
    float accum = 0.f;

    for (int i=0;i<n_int-1;++i){
        float t0=t_vals[i], t1=t_vals[i+1];
        if (isinf(t0)||isinf(t1)) continue;

        float x0=sx+t0*vx, y0=sy+t0*vy, z0=sz+t0*vz;
        float x1=sx+t1*vx, y1=sy+t1*vy, z1=sz+t1*vz;
        float seg_len = sqrtf((x1-x0)*(x1-x0)+(y1-y0)*(y1-y0)+(z1-z0)*(z1-z0));

        float mx=0.5f*(x0+x1), my=0.5f*(y0+y1), mz=0.5f*(z0+z1);
        float i_f,j_f,k_f;
        apply_affine_inverse_3d(mx,my,mz, M_inv,b, i_f,j_f,k_f);

        int i_i = __float2int_rn(i_f);
        int j_i = __float2int_rn(j_f);
        int k_i = __float2int_rn(k_f);
        // float dirx = dx - sx, diry = dy - sy, dirz = dz - sz;
        // int i_i = (dirx >= 0.f) ? (int)floorf(i_f) : (int)ceilf(i_f) - 1;
        // int j_i = (diry >= 0.f) ? (int)floorf(j_f) : (int)ceilf(j_f) - 1;
        // int k_i = (dirz >= 0.f) ? (int)floorf(k_f) : (int)ceilf(k_f) - 1;

        if (i_i<0||i_i>=n_x||j_i<0||j_i>=n_y||k_i<0||k_i>=n_z) continue;

        size_t idx = (size_t)b_idx*n_x*n_y*n_z
                   + (size_t)i_i*n_y*n_z
                   + (size_t)j_i*n_z
                   + (size_t)k_i;

        accum += volume[idx]*seg_len;
    }
    out[b_idx*n_ray + r_idx] = accum;
}

// ……………………… back_project_3d_kernel ………………………
__global__ void back_project_3d_kernel(
    float* out_vol,int batch,int n_x,int n_y,int n_z,
    const float* t_sorted,int n_int,
    const float* sino,int n_ray,
    const float* src_xyz,const float* dst_xyz,
    const float* M_inv,const float* b)
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int total = batch*n_ray;
    if (gid>=total) return;

    int b_idx = gid / n_ray;
    int r_idx = gid % n_ray;

    float sx=src_xyz[3*r_idx+0], sy=src_xyz[3*r_idx+1], sz=src_xyz[3*r_idx+2];
    float dx=dst_xyz[3*r_idx+0], dy=dst_xyz[3*r_idx+1], dz=dst_xyz[3*r_idx+2];
    float vx=dx-sx, vy=dy-sy, vz=dz-sz;

    const float* t_vals = &t_sorted[r_idx*n_int];
    float s_val = sino[b_idx*n_ray + r_idx];

    for (int i=0;i<n_int-1;++i){
        float t0=t_vals[i], t1=t_vals[i+1];
        if (isinf(t0)||isinf(t1)) continue;

        float x0=sx+t0*vx, y0=sy+t0*vy, z0=sz+t0*vz;
        float x1=sx+t1*vx, y1=sy+t1*vy, z1=sz+t1*vz;
        float seg_len = sqrtf((x1-x0)*(x1-x0)+(y1-y0)*(y1-y0)+(z1-z0)*(z1-z0));

        float mx=0.5f*(x0+x1), my=0.5f*(y0+y1), mz=0.5f*(z0+z1);
        float i_f,j_f,k_f;
        apply_affine_inverse_3d(mx,my,mz, M_inv,b, i_f,j_f,k_f);

        int i_i = __float2int_rn(i_f);
        int j_i = __float2int_rn(j_f);
        int k_i = __float2int_rn(k_f);
        // float dirx = dx - sx, diry = dy - sy, dirz = dz - sz;
        // int i_i = (dirx >= 0.f) ? (int)floorf(i_f) : (int)ceilf(i_f) - 1;
        // int j_i = (diry >= 0.f) ? (int)floorf(j_f) : (int)ceilf(j_f) - 1;
        // int k_i = (dirz >= 0.f) ? (int)floorf(k_f) : (int)ceilf(k_f) - 1;

        if (i_i<0||i_i>=n_x||j_i<0||j_i>=n_y||k_i<0||k_i>=n_z) continue;

        float contrib = s_val*seg_len;
        size_t idx = (size_t)b_idx*n_x*n_y*n_z
                   + (size_t)i_i*n_y*n_z
                   + (size_t)j_i*n_z
                   + (size_t)k_i;
        atomicAdd(&out_vol[idx], contrib);
    }
}

// ------------------------------------------------------------
// HOST wrappers – signatures unchanged, but they now expect M_inv
// ------------------------------------------------------------
torch::Tensor compute_intersections_3d(
    int64_t n_x,int64_t n_y,int64_t n_z,
    torch::Tensor src,torch::Tensor dst,
    torch::Tensor M_inv,torch::Tensor b)
{
    TORCH_CHECK(src.is_cuda() && dst.is_cuda(), "src/dst must be CUDA");
    TORCH_CHECK(M_inv.is_cuda() && b.is_cuda(), "M_inv,b must be CUDA");

    int64_t n_ray = src.size(0);
    int64_t n_int = (n_x+1)+(n_y+1)+(n_z+1);
    auto out = torch::empty({n_ray,n_int}, src.options().dtype(torch::kFloat32));

    int threads = THREADS;
    int blocks  = (int)((n_ray+threads-1)/threads);
    compute_intersections_3d_kernel<<<blocks,threads>>>(
        (int)n_x,(int)n_y,(int)n_z,
        src.data_ptr<float>(),dst.data_ptr<float>(),(int)n_ray,
        M_inv.data_ptr<float>(),b.data_ptr<float>(),
        out.data_ptr<float>());
    return out;
}

torch::Tensor forward_project_3d_cuda(
    torch::Tensor volume,torch::Tensor tvals,
    torch::Tensor src,torch::Tensor dst,
    torch::Tensor M_inv,torch::Tensor b)
{
    TORCH_CHECK(volume.is_cuda() && tvals.is_cuda(),"volume/tvals must be CUDA");

    int64_t batch=1,n_x,n_y,n_z;
    if (volume.dim()==3){n_x=volume.size(0);n_y=volume.size(1);n_z=volume.size(2);}
    else if (volume.dim()==4){
        batch=volume.size(0);n_x=volume.size(1);n_y=volume.size(2);n_z=volume.size(3);}
    else TORCH_CHECK(false,"volume shape!");

    int64_t n_ray = src.size(0);
    int64_t n_int = tvals.size(1);
    auto out = torch::zeros({batch,n_ray}, volume.options());

    int threads=THREADS, blocks=(int)((batch*n_ray+threads-1)/threads);
    forward_project_3d_kernel<<<blocks,threads>>>(
        volume.data_ptr<float>(),(int)batch,(int)n_x,(int)n_y,(int)n_z,
        tvals.data_ptr<float>(),(int)n_int,
        src.data_ptr<float>(),dst.data_ptr<float>(),(int)n_ray,
        M_inv.data_ptr<float>(),b.data_ptr<float>(),
        out.data_ptr<float>());
    return out;
}

torch::Tensor back_project_3d_cuda(
    torch::Tensor sino,torch::Tensor tvals,
    torch::Tensor src,torch::Tensor dst,
    torch::Tensor M_inv,torch::Tensor b,
    int64_t n_x,int64_t n_y,int64_t n_z)
{
    TORCH_CHECK(sino.is_cuda() && tvals.is_cuda(),"sino/tvals must be CUDA");

    int64_t batch=1,n_ray;
    if (sino.dim()==1){n_ray=sino.size(0);}
    else if (sino.dim()==2){batch=sino.size(0);n_ray=sino.size(1);}
    else TORCH_CHECK(false,"sino shape!");

    int64_t n_int = tvals.size(1);
    auto out     = torch::zeros({batch,n_x,n_y,n_z}, sino.options());

    int threads=THREADS, blocks=(int)((batch*n_ray+threads-1)/threads);
    back_project_3d_kernel<<<blocks,threads>>>(
        out.data_ptr<float>(),(int)batch,(int)n_x,(int)n_y,(int)n_z,
        tvals.data_ptr<float>(),(int)n_int,
        sino.data_ptr<float>(),(int)n_ray,
        src.data_ptr<float>(),dst.data_ptr<float>(),
        M_inv.data_ptr<float>(),b.data_ptr<float>());
    return out;
}

// /* ========================================================================== */
// /*            S I D D O N   O N - T H E - F L Y   (shared helpers)           */
// /* ========================================================================== */

// struct RayState {
//     int   ii, jj, kk;
//     int   step_i, step_j, step_k;      //  NEW
//     float t_cur, t_exit;
//     float t_next_i, t_next_j, t_next_k;
//     float dt_i, dt_j, dt_k;
//     float ray_len;
// };
// __device__ bool siddon_initialise(
//     int n_x, int n_y, int n_z,
//     float sx, float sy, float sz,
//     float dx, float dy, float dz,
//     const float* M_inv, const float* b,
//     RayState& st)
// {
//     // physical ray length
//     float vx = dx - sx, vy = dy - sy, vz = dz - sz;
//     st.ray_len = sqrtf(vx*vx + vy*vy + vz*vz);
//     if (st.ray_len < 1e-12f) return false;

//     // world → voxel endpoints
//     float si, sj, sk, di, dj, dk;
//     apply_affine_inverse_3d(sx, sy, sz, M_inv, b, si, sj, sk);
//     apply_affine_inverse_3d(dx, dy, dz, M_inv, b, di, dj, dk);

//     // direction in voxel space
//     float vi = di - si, vj = dj - sj, vk = dk - sk;

//     // record whether each axis is increasing or decreasing
//     st.step_i = (vi >= 0.f) ? +1 : -1;
//     st.step_j = (vj >= 0.f) ? +1 : -1;
//     st.step_k = (vk >= 0.f) ? +1 : -1;

//     // reciprocals for parametric t
//     float inv_vi = (fabsf(vi) < 1e-12f) ? 1e32f : 1.f/vi;
//     float inv_vj = (fabsf(vj) < 1e-12f) ? 1e32f : 1.f/vj;
//     float inv_vk = (fabsf(vk) < 1e-12f) ? 1e32f : 1.f/vk;

//     // entry/exit parameters
//     float t_i0 = (-0.5f         - si) * inv_vi;
//     float t_i1 = ((float)n_x-0.5f - si) * inv_vi;
//     float t_j0 = (-0.5f         - sj) * inv_vj;
//     float t_j1 = ((float)n_y-0.5f - sj) * inv_vj;
//     float t_k0 = (-0.5f         - sk) * inv_vk;
//     float t_k1 = ((float)n_z-0.5f - sk) * inv_vk;

//     st.t_cur  = fmaxf(fmaxf(fminf(t_i0,t_i1), fminf(t_j0,t_j1)),
//                       fminf(t_k0,t_k1));
//     st.t_exit = fminf(fminf(fmaxf(t_i0,t_i1), fmaxf(t_j0,t_j1)),
//                       fmaxf(t_k0,t_k1));
//     if (st.t_exit <= st.t_cur) return false;

//     // clamp to [0,1]
//     st.t_cur  = fmaxf(st.t_cur,  0.f);
//     st.t_exit = fminf(st.t_exit, 1.f);

//     // compute entry-point coordinates
//     float i_ent = si + st.t_cur * vi;
//     float j_ent = sj + st.t_cur * vj;
//     float k_ent = sk + st.t_cur * vk;

//     // first voxel indices
//     st.ii = (vi >= 0.f) ? (int)floorf(i_ent) : (int)ceilf(i_ent) - 1;
//     st.jj = (vj >= 0.f) ? (int)floorf(j_ent) : (int)ceilf(j_ent) - 1;
//     st.kk = (vk >= 0.f) ? (int)floorf(k_ent) : (int)ceilf(k_ent) - 1;

//     // time of next boundary crossing
//     st.t_next_i = (vi >= 0.f)
//         ? st.t_cur + ((st.ii + 0.5f) - i_ent) * inv_vi
//         : st.t_cur + ((st.ii - 0.5f) - i_ent) * inv_vi;
//     st.t_next_j = (vj >= 0.f)
//         ? st.t_cur + ((st.jj + 0.5f) - j_ent) * inv_vj
//         : st.t_cur + ((st.jj - 0.5f) - j_ent) * inv_vj;
//     st.t_next_k = (vk >= 0.f)
//         ? st.t_cur + ((st.kk + 0.5f) - k_ent) * inv_vk
//         : st.t_cur + ((st.kk - 0.5f) - k_ent) * inv_vk;

//     // absolute step sizes
//     st.dt_i = fabsf(inv_vi);
//     st.dt_j = fabsf(inv_vj);
//     st.dt_k = fabsf(inv_vk);

//     return true;
// }

// template <typename Op>
// __device__ void siddon_march(
//     int n_x, int n_y, int n_z,
//     RayState st, Op& op)
// {
//     while (st.t_cur < st.t_exit)
//     {
//         float t_hit;
//         int axis;
//         if (st.t_next_i <= st.t_next_j && st.t_next_i <= st.t_next_k) {
//             t_hit = st.t_next_i; axis = 0;
//         } else if (st.t_next_j <= st.t_next_i && st.t_next_j <= st.t_next_k) {
//             t_hit = st.t_next_j; axis = 1;
//         } else {
//             t_hit = st.t_next_k; axis = 2;
//         }
//         t_hit = fminf(t_hit, st.t_exit);

//         if (st.ii>=0 && st.ii<n_x &&
//             st.jj>=0 && st.jj<n_y &&
//             st.kk>=0 && st.kk<n_z)
//         {
//             op.add(st.ii, st.jj, st.kk, (t_hit - st.t_cur)*st.ray_len);
//         }

//         st.t_cur = t_hit;
//         if      (axis == 0){
//             st.t_next_i += st.dt_i;
//             st.ii       += st.step_i;          //  FIX
//         }
//         else if (axis == 1){
//             st.t_next_j += st.dt_j;
//             st.jj       += st.step_j;          //  FIX
//         }
//         else {
//             st.t_next_k += st.dt_k;
//             st.kk       += st.step_k;          //  FIX
//         }
//     }
// }

// /* ========================================================================== */
// /*               K E R N E L S   (uses the shared helpers)                   */
// /* ========================================================================== */

// __global__ void forward_project_3d_on_the_fly_kernel(
//     const float* vol, int batch, int n_x,int n_y,int n_z,
//     const float* src_xyz,const float* dst_xyz,int n_ray,
//     const float* M_inv,const float* b,
//     float* out)
// {
//     int gid = blockIdx.x*blockDim.x + threadIdx.x;
//     if (gid >= batch*n_ray) return;
//     int bi = gid / n_ray, ri = gid % n_ray;
//     RayState st;
//     if (!siddon_initialise(n_x,n_y,n_z,
//           src_xyz[3*ri+0],src_xyz[3*ri+1],src_xyz[3*ri+2],
//           dst_xyz[3*ri+0],dst_xyz[3*ri+1],dst_xyz[3*ri+2],
//           M_inv,b, st)) {
//       out[gid] = 0.f; return;
//     }

//     struct Accum {
//       const float* v; int ny,nz; float sum;
//       __device__ void add(int i,int j,int k,float seg){
//         size_t idx = (size_t)i*ny*nz + (size_t)j*nz + k;
//         sum += v[idx]*seg;
//       }
//     } op{vol + (size_t)bi*n_x*n_y*n_z, n_y, n_z, 0.f};

//     siddon_march(n_x,n_y,n_z, st, op);
//     out[gid] = op.sum;
// }

// __global__ void back_project_3d_on_the_fly_kernel(
//     float* vol_out, int batch, int n_x,int n_y,int n_z,
//     const float* sino,int n_ray,
//     const float* src_xyz,const float* dst_xyz,
//     const float* M_inv,const float* b)
// {
//     int gid = blockIdx.x*blockDim.x + threadIdx.x;
//     if (gid >= batch*n_ray) return;
//     int bi = gid / n_ray, ri = gid % n_ray;
//     RayState st;
//     if (!siddon_initialise(n_x,n_y,n_z,
//           src_xyz[3*ri+0],src_xyz[3*ri+1],src_xyz[3*ri+2],
//           dst_xyz[3*ri+0],dst_xyz[3*ri+1],dst_xyz[3*ri+2],
//           M_inv,b, st)) return;

//     float val = sino[gid];
//     struct Accum {
//       float* v; int ny,nz; float w;
//       __device__ void add(int i,int j,int k,float seg){
//         size_t idx = (size_t)i*ny*nz + (size_t)j*nz + k;
//         atomicAdd(&v[idx], w*seg);
//       }
//     } op{vol_out + (size_t)bi*n_x*n_y*n_z, n_y, n_z, val};

//     siddon_march(n_x,n_y,n_z, st, op);
// }

// /* ========================================================================== */
// /*            H O S T   W R A P P E R S   (exported entry points)            */
// /* ========================================================================== */

// torch::Tensor forward_project_3d_on_the_fly_cuda(
//     torch::Tensor volume,
//     torch::Tensor src, torch::Tensor dst,
//     torch::Tensor M_inv, torch::Tensor b)
// {
//     TORCH_CHECK(volume.is_cuda() && src.is_cuda() && dst.is_cuda(),
//                 "inputs must be CUDA");
//     int64_t batch = 1, n_x,n_y,n_z;
//     if (volume.dim()==3) {
//       n_x=volume.size(0); n_y=volume.size(1); n_z=volume.size(2);
//     } else {
//       batch=volume.size(0);
//       n_x=volume.size(1); n_y=volume.size(2); n_z=volume.size(3);
//     }
//     int64_t n_ray = src.size(0);
//     auto out = torch::zeros({batch,n_ray}, volume.options());
//     int threads=THREADS, blocks=(int)((batch*n_ray+threads-1)/threads);
//     forward_project_3d_on_the_fly_kernel<<<blocks,threads>>>(
//       volume.data_ptr<float>(),
//       (int)batch,(int)n_x,(int)n_y,(int)n_z,
//       src.data_ptr<float>(), dst.data_ptr<float>(), (int)n_ray,
//       M_inv.data_ptr<float>(), b.data_ptr<float>(),
//       out.data_ptr<float>());
//     return out;
// }

// torch::Tensor back_project_3d_on_the_fly_cuda(
//     torch::Tensor sino,
//     torch::Tensor src, torch::Tensor dst,
//     torch::Tensor M_inv, torch::Tensor b,
//     int64_t n_x,int64_t n_y,int64_t n_z)
// {
//     TORCH_CHECK(sino.is_cuda() && src.is_cuda() && dst.is_cuda(),
//                 "inputs must be CUDA");
//     int64_t batch=1, n_ray;
//     if (sino.dim()==1)         n_ray=sino.size(0);
//     else /* dim==2 */          { batch=sino.size(0); n_ray=sino.size(1); }
//     auto out = torch::zeros({batch,n_x,n_y,n_z}, sino.options());
//     int threads=THREADS, blocks=(int)((batch*n_ray+threads-1)/threads);
//     back_project_3d_on_the_fly_kernel<<<blocks,threads>>>(
//       out.data_ptr<float>(),
//       (int)batch,(int)n_x,(int)n_y,(int)n_z,
//       sino.data_ptr<float>(), (int)n_ray,
//       src.data_ptr<float>(), dst.data_ptr<float>(),
//       M_inv.data_ptr<float>(), b.data_ptr<float>());
//     return out;
// }

























/* ========================================================================== */
/*     P L A N E - S O R T   O N - T H E - F L Y   (single-ray version)      */
/*            – computes ALL plane hits, sorts once, then marches –          */
/* ========================================================================== */

#ifndef MAX_INTERSECTIONS
// n_int = (n_x+1)+(n_y+1)+(n_z+1). 2048 covers 512³ volumes with margin.
#define MAX_INTERSECTIONS  2048
#endif

/* --------------------------------------------
   small insertion sort – works well for ≤2 k
   -------------------------------------------- */
__device__ __forceinline__
void sort_in_place(float* arr, int n)
{
    for (int i = 1; i < n; ++i){
        float key = arr[i];
        int   j   = i - 1;
        while (j >= 0 && arr[j] > key){
            arr[j+1] = arr[j];
            --j;
        }
        arr[j+1] = key;
    }
}

/* -------------------------------------------------
   gather all plane hits, return count (≤MAX_INTERSECTIONS)
   t is parametric distance:  P(t)=src + t*(dst-src),  t∈[0,1]
   ------------------------------------------------- */
__device__ __forceinline__
int collect_hits(
    int n_x,int n_y,int n_z,
    float si,float sj,float sk,
    float vi,float vj,float vk,
    const float* __restrict__ xplanes,
    const float* __restrict__ yplanes,
    const float* __restrict__ zplanes,
    float* t_buf)
{
    int cnt = 0;

    // X planes
    if (fabsf(vi) > 1e-12f){
        float inv_vi = 1.f/vi;
        for (int i = 0; i <= n_x; ++i){
            float t = (xplanes[i] - si) * inv_vi;
            if (t >= 0.f && t <= 1.f) t_buf[cnt++] = t;
        }
    }
    // Y planes
    if (fabsf(vj) > 1e-12f){
        float inv_vj = 1.f/vj;
        for (int j = 0; j <= n_y; ++j){
            float t = (yplanes[j] - sj) * inv_vj;
            if (t >= 0.f && t <= 1.f) t_buf[cnt++] = t;
        }
    }
    // Z planes
    if (fabsf(vk) > 1e-12f){
        float inv_vk = 1.f/vk;
        for (int k = 0; k <= n_z; ++k){
            float t = (zplanes[k] - sk) * inv_vk;
            if (t >= 0.f && t <= 1.f) t_buf[cnt++] = t;
        }
    }

    // ensure end-points are present
    t_buf[cnt++] = 0.f;
    t_buf[cnt++] = 1.f;

    sort_in_place(t_buf, cnt);
    return cnt;
}

/* =========================================================
   kernels – one thread = one ray
   Shared memory layout:   [ X-planes | Y-planes | Z-planes ]
   ========================================================= */

template <bool BACKWARD>
__global__ void ray_kernel_plane_sort(
    /* volume / sino pointers                                  */
    float* __restrict__ vol_or_out,
    const float* __restrict__ vol_or_sino,
    /* sizes */
    int batch,int n_x,int n_y,int n_z,
    /* rays */
    const float* __restrict__ src_xyz,
    const float* __restrict__ dst_xyz,
    int n_ray,
    /* voxel-space transform */
    const float* __restrict__ M_inv,
    const float* __restrict__ b)
{
    /* shared memory for the voxel-plane coordinates */
    extern __shared__ float s_planes[];
    float* xplanes = s_planes;                  // size n_x+1
    float* yplanes = xplanes + (n_x+1);         // size n_y+1
    float* zplanes = yplanes + (n_y+1);         // size n_z+1

    /* load planes once per block */
    for (int idx = threadIdx.x; idx <= n_x; idx += blockDim.x)
        xplanes[idx] = float(idx) - 0.5f;
    for (int idx = threadIdx.x; idx <= n_y; idx += blockDim.x)
        yplanes[idx] = float(idx) - 0.5f;
    for (int idx = threadIdx.x; idx <= n_z; idx += blockDim.x)
        zplanes[idx] = float(idx) - 0.5f;
    __syncthreads();

    /* which ray? */
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if (gid >= batch*n_ray) return;
    int bi = gid / n_ray;
    int ri = gid % n_ray;

    /* world coordinates */
    float sx = src_xyz[3*ri+0], sy = src_xyz[3*ri+1], sz = src_xyz[3*ri+2];
    float dx = dst_xyz[3*ri+0], dy = dst_xyz[3*ri+1], dz = dst_xyz[3*ri+2];
    float vx = dx - sx, vy = dy - sy, vz = dz - sz;
    float ray_len = sqrtf(vx*vx + vy*vy + vz*vz);
    if (ray_len < 1e-12f) {
        if constexpr(!BACKWARD) vol_or_out[gid] = 0.f;
        return;
    }

    /* endpoints in voxel space */
    float si,sj,sk, di,dj,dk;
    apply_affine_inverse_3d(sx,sy,sz,M_inv,b, si,sj,sk);
    apply_affine_inverse_3d(dx,dy,dz,M_inv,b, di,dj,dk);
    float vi = di-si, vj = dj-sj, vk = dk-sk;

    /* local buffer for t’s */
    float t_vals[MAX_INTERSECTIONS];
    int n_t = collect_hits(n_x,n_y,n_z,
                           si,sj,sk, vi,vj,vk,
                           xplanes,yplanes,zplanes,
                           t_vals);

    /* choose correct base pointers */
    const float* vol  = nullptr;
          float* volO = nullptr;
    const float* sino = nullptr;
    if constexpr(!BACKWARD){
        vol  = vol_or_sino + (size_t)bi*n_x*n_y*n_z;
    }else{
        volO = vol_or_out  + (size_t)bi*n_x*n_y*n_z;
        sino = vol_or_sino;
    }

    /* accumulator */
    float fp_accum = 0.f;
    float val      = BACKWARD ? sino[gid] : 0.f;

    /* iterate segments */
    for (int idx = 0; idx < n_t-1; ++idx){
        float t0 = t_vals[idx];
        float t1 = t_vals[idx+1];
        if (t1 <= t0) continue;   // zero length

        float t_mid = 0.5f*(t0+t1);

        /* midpoint in world for voxel lookup */
        float mx = sx + t_mid*vx;
        float my = sy + t_mid*vy;
        float mz = sz + t_mid*vz;

        /* voxel index */
        float i_f,j_f,k_f;
        apply_affine_inverse_3d(mx,my,mz,M_inv,b, i_f,j_f,k_f);
        int   ii = __float2int_rn(i_f);
        int   jj = __float2int_rn(j_f);
        int   kk = __float2int_rn(k_f);

        if (ii<0||ii>=n_x||jj<0||jj>=n_y||kk<0||kk>=n_z) continue;

        float seg_len = (t1-t0)*ray_len;

        size_t v_idx = (size_t)ii*n_y*n_z + (size_t)jj*n_z + kk;

        if constexpr(!BACKWARD){
            fp_accum += vol[v_idx]*seg_len;
        }else{
            atomicAdd(&volO[v_idx], val*seg_len);
        }
    }

    if constexpr(!BACKWARD){
        vol_or_out[gid] = fp_accum;
    }
}

/* ========================================
   H O S T   W R A P P E R S  (unchanged api)
   ======================================== */

torch::Tensor forward_project_3d_on_the_fly_cuda(
    torch::Tensor volume,
    torch::Tensor src, torch::Tensor dst,
    torch::Tensor M_inv, torch::Tensor b)
{
    TORCH_CHECK(volume.is_cuda() && src.is_cuda() && dst.is_cuda(),
                "inputs must be CUDA");

    int64_t batch = 1, n_x,n_y,n_z;
    if (volume.dim()==3){
        n_x=volume.size(0); n_y=volume.size(1); n_z=volume.size(2);
    } else {
        batch=volume.size(0);
        n_x=volume.size(1); n_y=volume.size(2); n_z=volume.size(3);
    }
    int64_t n_ray = src.size(0);
    auto out = torch::zeros({batch,n_ray}, volume.options());

    int threads = THREADS;
    int blocks  = (int)((batch*n_ray + threads - 1) / threads);
    size_t smem = (n_x+1 + n_y+1 + n_z+1) * sizeof(float);

    ray_kernel_plane_sort<false><<<blocks,threads,smem>>>(
        /* vol_or_out  */ out.data_ptr<float>(),
        /* vol_or_sino */ volume.data_ptr<float>(),
        (int)batch,(int)n_x,(int)n_y,(int)n_z,
        src.data_ptr<float>(), dst.data_ptr<float>(),
        (int)n_ray,
        M_inv.data_ptr<float>(), b.data_ptr<float>());

    return out;
}

torch::Tensor back_project_3d_on_the_fly_cuda(
    torch::Tensor sino,
    torch::Tensor src, torch::Tensor dst,
    torch::Tensor M_inv, torch::Tensor b,
    int64_t n_x,int64_t n_y,int64_t n_z)
{
    TORCH_CHECK(sino.is_cuda() && src.is_cuda() && dst.is_cuda(),
                "inputs must be CUDA");

    int64_t batch = 1, n_ray;
    if (sino.dim()==1)              n_ray = sino.size(0);
    else /* dim==2 */              { batch = sino.size(0); n_ray = sino.size(1); }

    auto out = torch::zeros({batch,n_x,n_y,n_z}, sino.options());

    int threads = THREADS;
    int blocks  = (int)((batch*n_ray + threads - 1) / threads);
    size_t smem = (n_x+1 + n_y+1 + n_z+1) * sizeof(float);

    ray_kernel_plane_sort<true><<<blocks,threads,smem>>>(
        /* vol_or_out  */ out.data_ptr<float>(),
        /* vol_or_sino */ sino.data_ptr<float>(),
        (int)batch,(int)n_x,(int)n_y,(int)n_z,
        src.data_ptr<float>(), dst.data_ptr<float>(),
        (int)n_ray,
        M_inv.data_ptr<float>(), b.data_ptr<float>());

    return out;
}